#include "hip/hip_runtime.h"
#include "cuda_pi.h"

const double PI =  3.141592653589793;
double base = 0.0;

// Se utiliza un array donde se guardan los resultados parciales
// de cada streaming-thread
double partials[TH_COUNT];
double *gpu_partials;

__global__ void calc_pi_th(
    double *partial_results,
    long long int num_iteraciones,
    double base)
{    
    // indice global del thread (1D)
    int th_id = blockIdx.x * blockDim.x + threadIdx.x;
    // Se desea manejar cantidad enormes de intervalos
    long long int i = 0;
    // Valores para calculo de PI
    double acum = 0, x;
    // Se asume que la cantidad de intervalos es multiplo
    // de la cantidad de threads simultaneos.
    // Se inicia a calcular acorde al indice del hilo.
    x = base * (th_id) * num_iteraciones;    
    #pragma unroll
    for(i = 0; i < num_iteraciones; i++) {
        acum += 4/(1 + x * x);
        x += base;
    }

    partial_results[th_id] = acum;
}

int main(int argc, char const *argv[])
{    
    // cuda malloc
    hipMalloc((void**)&gpu_partials, sizeof(double)*TH_COUNT);
    /* 245,760 */
    run_pi_calc(245760);
    /* 24,760,000 */
    run_pi_calc(24576000);
    /* 2,457,600,000 */
    run_pi_calc(2457600000);
    // cuda free
    hipFree(gpu_partials);
    hipDeviceReset();
}

void run_pi_calc(long long int intervalos)
{        
    double pi = 0.0;
    double elapsed;
    base = 1.0/intervalos;

    StartTimer();    
    printf("\nCalculando pi con %lld intervalos\n", intervalos);    

    map_pi_calc(intervalos, base);
    pi = reduce_pi_calc()*base;

    elapsed = GetTimer();
    printf("PI=%1.15lf\n", pi);
    printf("Error: %1.15lf\n", PI - pi);
    printf("Tiempo: %4.2lf ms\n", elapsed);
}

// maps the pi calculation to each device
void map_pi_calc(long long int intervalos, double base)
{
    long long int intervalo_count = intervalos / TH_COUNT;    
    // ejecutar calculo    
    calc_pi_th<<< GRID_SIZE, BLOCK_SIZE >>>(gpu_partials, intervalo_count, base);
    // memcpy gpu -> cpu
    hipMemcpy(partials, gpu_partials, sizeof(double)*TH_COUNT, hipMemcpyDeviceToHost);
}

// integrates the results of all the streaming threads
double reduce_pi_calc()
{
    double acum = 0;
    int i = 0;
    for(i = 0; i < TH_COUNT; i++) {
        acum += partials[i];
    }

    return acum;
}